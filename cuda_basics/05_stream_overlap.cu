#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

// 简单的计算密集型内核
__global__ void computeIntensive(float *input, float *output, int n, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float val = input[idx];
        // 执行一些计算密集的操作
        for (int i = 0; i < iterations; i++) {
            val = sinf(val) + cosf(val);
        }
        output[idx] = val;
    }
}

// 向量加法内核
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// 测量时间的辅助函数
float measureTime(hipEvent_t start, hipEvent_t stop) {
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
}

// 初始化数据
void initializeData(float *data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = (float)(rand() % 100) / 100.0f;
    }
}

// 无重叠的同步执行
float sequentialExecution(float *h_input, float *h_output, int dataSize, int numStreams) {
    const int blockSize = 256;
    const int segmentSize = dataSize / numStreams;
    
    float *d_input, *d_output;
    hipMalloc(&d_input, dataSize * sizeof(float));
    hipMalloc(&d_output, dataSize * sizeof(float));
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    for (int i = 0; i < numStreams; i++) {
        int offset = i * segmentSize;
        int currentSegmentSize = (i == numStreams - 1) ? dataSize - offset : segmentSize;
        
        // 数据传输到设备
        hipMemcpy(d_input + offset, h_input + offset, 
                   currentSegmentSize * sizeof(float), hipMemcpyHostToDevice);
        
        // 执行计算
        int gridSize = (currentSegmentSize + blockSize - 1) / blockSize;
        computeIntensive<<<gridSize, blockSize>>>(d_input + offset, d_output + offset, 
                                                 currentSegmentSize, 1000);
        
        // 数据传输回主机
        hipMemcpy(h_output + offset, d_output + offset, 
                   currentSegmentSize * sizeof(float), hipMemcpyDeviceToHost);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float elapsed = measureTime(start, stop);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_input);
    hipFree(d_output);
    
    return elapsed;
}

// 使用多个流进行重叠执行
float overlappedExecution(float *h_input, float *h_output, int dataSize, int numStreams) {
    const int blockSize = 256;
    const int segmentSize = dataSize / numStreams;
    
    // 分配设备内存
    float *d_input, *d_output;
    hipMalloc(&d_input, dataSize * sizeof(float));
    hipMalloc(&d_output, dataSize * sizeof(float));
    
    // 分配页锁定主机内存（提高传输性能）
    float *h_input_pinned, *h_output_pinned;
    hipHostMalloc(&h_input_pinned, dataSize * sizeof(float));
    hipHostMalloc(&h_output_pinned, dataSize * sizeof(float));
    
    // 复制数据到页锁定内存
    memcpy(h_input_pinned, h_input, dataSize * sizeof(float));
    
    // 创建多个流
    hipStream_t *streams = (hipStream_t*)malloc(numStreams * sizeof(hipStream_t));
    for (int i = 0; i < numStreams; i++) {
        hipStreamCreate(&streams[i]);
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    // 启动所有流的操作
    for (int i = 0; i < numStreams; i++) {
        int offset = i * segmentSize;
        int currentSegmentSize = (i == numStreams - 1) ? dataSize - offset : segmentSize;
        
        // 异步内存传输到设备
        hipMemcpyAsync(d_input + offset, h_input_pinned + offset, 
                        currentSegmentSize * sizeof(float), 
                        hipMemcpyHostToDevice, streams[i]);
        
        // 异步执行内核
        int gridSize = (currentSegmentSize + blockSize - 1) / blockSize;
        computeIntensive<<<gridSize, blockSize, 0, streams[i]>>>(
            d_input + offset, d_output + offset, currentSegmentSize, 1000);
        
        // 异步内存传输回主机
        hipMemcpyAsync(h_output_pinned + offset, d_output + offset, 
                        currentSegmentSize * sizeof(float), 
                        hipMemcpyDeviceToHost, streams[i]);
    }
    
    // 等待所有流完成
    for (int i = 0; i < numStreams; i++) {
        hipStreamSynchronize(streams[i]);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float elapsed = measureTime(start, stop);
    
    // 复制结果回原始内存
    memcpy(h_output, h_output_pinned, dataSize * sizeof(float));
    
    // 清理资源
    for (int i = 0; i < numStreams; i++) {
        hipStreamDestroy(streams[i]);
    }
    free(streams);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipHostFree(h_input_pinned);
    hipHostFree(h_output_pinned);
    hipFree(d_input);
    hipFree(d_output);
    
    return elapsed;
}

// 演示流优先级的使用
void demonstratePriority() {
    printf("\n=== 演示流优先级 ===\n");
    
    const int dataSize = 1024 * 1024;
    const int blockSize = 256;
    const int gridSize = (dataSize + blockSize - 1) / blockSize;
    
    // 分配内存
    float *d_data1, *d_data2, *d_result1, *d_result2;
    hipMalloc(&d_data1, dataSize * sizeof(float));
    hipMalloc(&d_data2, dataSize * sizeof(float));
    hipMalloc(&d_result1, dataSize * sizeof(float));
    hipMalloc(&d_result2, dataSize * sizeof(float));
    
    // 创建不同优先级的流
    hipStream_t highPriorityStream, lowPriorityStream;
    
    int leastPriority, greatestPriority;
    hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
    
    hipStreamCreateWithPriority(&highPriorityStream, hipStreamNonBlocking, greatestPriority);
    hipStreamCreateWithPriority(&lowPriorityStream, hipStreamNonBlocking, leastPriority);
    
    printf("优先级范围: %d (最低) 到 %d (最高)\n", leastPriority, greatestPriority);
    
    // 创建事件来测量时间
    hipEvent_t start1, start2, stop1, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);
    
    // 启动低优先级任务
    hipEventRecord(start2, lowPriorityStream);
    computeIntensive<<<gridSize, blockSize, 0, lowPriorityStream>>>(d_data2, d_result2, dataSize, 2000);
    hipEventRecord(stop2, lowPriorityStream);
    
    // 稍后启动高优先级任务
    hipEventRecord(start1, highPriorityStream);
    computeIntensive<<<gridSize, blockSize, 0, highPriorityStream>>>(d_data1, d_result1, dataSize, 1000);
    hipEventRecord(stop1, highPriorityStream);
    
    // 等待完成并测量时间
    hipEventSynchronize(stop1);
    hipEventSynchronize(stop2);
    
    float time1 = measureTime(start1, stop1);
    float time2 = measureTime(start2, stop2);
    
    printf("高优先级流执行时间: %.2f ms\n", time1);
    printf("低优先级流执行时间: %.2f ms\n", time2);
    
    // 清理
    hipStreamDestroy(highPriorityStream);
    hipStreamDestroy(lowPriorityStream);
    hipEventDestroy(start1);
    hipEventDestroy(start2);
    hipEventDestroy(stop1);
    hipEventDestroy(stop2);
    hipFree(d_data1);
    hipFree(d_data2);
    hipFree(d_result1);
    hipFree(d_result2);
}

// 演示多流计算和传输重叠
void demonstrateComputeTransferOverlap() {
    printf("\n=== 计算和传输重叠演示 ===\n");
    
    const int dataSize = 2 * 1024 * 1024; // 2M 元素
    const int numBatches = 4;
    const int batchSize = dataSize / numBatches;
    const int blockSize = 256;
    
    // 分配页锁定主机内存
    float *h_input, *h_output;
    hipHostMalloc(&h_input, dataSize * sizeof(float));
    hipHostMalloc(&h_output, dataSize * sizeof(float));
    
    // 分配设备内存
    float *d_input, *d_output;
    hipMalloc(&d_input, dataSize * sizeof(float));
    hipMalloc(&d_output, dataSize * sizeof(float));
    
    // 初始化数据
    initializeData(h_input, dataSize);
    
    // 创建流
    hipStream_t computeStream, transferStream;
    hipStreamCreate(&computeStream);
    hipStreamCreate(&transferStream);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    printf("处理 %d 个批次，每批次 %d 个元素\n", numBatches, batchSize);
    
    hipEventRecord(start);
    
    for (int batch = 0; batch < numBatches; batch++) {
        int offset = batch * batchSize;
        int gridSize = (batchSize + blockSize - 1) / blockSize;
        
        // 数据传输到设备（使用传输流）
        hipMemcpyAsync(d_input + offset, h_input + offset, 
                        batchSize * sizeof(float), 
                        hipMemcpyHostToDevice, transferStream);
        
        // 等待传输完成后开始计算
        hipStreamWaitEvent(computeStream, 0, 0);
        
        // 执行计算（使用计算流）
        computeIntensive<<<gridSize, blockSize, 0, computeStream>>>(
            d_input + offset, d_output + offset, batchSize, 1500);
        
        // 计算完成后传输结果回主机
        hipMemcpyAsync(h_output + offset, d_output + offset, 
                        batchSize * sizeof(float), 
                        hipMemcpyDeviceToHost, transferStream);
    }
    
    // 同步所有操作
    hipStreamSynchronize(computeStream);
    hipStreamSynchronize(transferStream);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float elapsed = measureTime(start, stop);
    printf("重叠执行总时间: %.2f ms\n", elapsed);
    
    // 清理
    hipStreamDestroy(computeStream);
    hipStreamDestroy(transferStream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipHostFree(h_input);
    hipHostFree(h_output);
    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    printf("CUDA Stream 重叠技术演示\n");
    printf("========================\n");
    
    // 检查设备能力
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("设备: %s\n", prop.name);
    printf("支持并发内核执行: %s\n", prop.concurrentKernels ? "是" : "否");
    printf("支持重叠数据传输: %s\n", prop.deviceOverlap ? "是" : "否");
    printf("异步引擎数量: %d\n", prop.asyncEngineCount);
    printf("\n");
    
    const int dataSize = 4 * 1024 * 1024; // 4M 元素
    const int numStreams = 4;
    
    // 分配主机内存
    float *h_input = (float*)malloc(dataSize * sizeof(float));
    float *h_output1 = (float*)malloc(dataSize * sizeof(float));
    float *h_output2 = (float*)malloc(dataSize * sizeof(float));
    
    // 初始化数据
    initializeData(h_input, dataSize);
    
    printf("=== 性能比较：同步 vs 异步执行 ===\n");
    printf("数据大小: %d 元素\n", dataSize);
    printf("流数量: %d\n\n", numStreams);
    
    // 测试同步执行
    printf("1. 同步执行（无重叠）...\n");
    float seqTime = sequentialExecution(h_input, h_output1, dataSize, numStreams);
    printf("同步执行时间: %.2f ms\n\n", seqTime);
    
    // 测试异步重叠执行
    printf("2. 异步执行（重叠）...\n");
    float overlapTime = overlappedExecution(h_input, h_output2, dataSize, numStreams);
    printf("重叠执行时间: %.2f ms\n", overlapTime);
    
    // 计算加速比
    float speedup = seqTime / overlapTime;
    printf("加速比: %.2fx\n", speedup);
    printf("性能提升: %.1f%%\n", (speedup - 1) * 100);
    
    // 验证结果一致性
    bool resultsMatch = true;
    for (int i = 0; i < dataSize && resultsMatch; i++) {
        if (fabs(h_output1[i] - h_output2[i]) > 1e-5) {
            resultsMatch = false;
        }
    }
    printf("结果验证: %s\n", resultsMatch ? "一致" : "不一致");
    
    // 演示其他Stream技术
    demonstratePriority();
    demonstrateComputeTransferOverlap();
    
    printf("\n=== Stream重叠优化总结 ===\n");
    printf("1. 使用页锁定内存提高传输性能\n");
    printf("2. 多流并行可以重叠计算和数据传输\n");
    printf("3. 流优先级可以控制任务调度\n");
    printf("4. 合理分批处理可以隐藏传输延迟\n");
    printf("5. 事件同步可以精确控制流间依赖关系\n");
    
    // 清理内存
    free(h_input);
    free(h_output1);
    free(h_output2);
    
    return 0;
} 